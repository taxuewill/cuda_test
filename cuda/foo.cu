#include "hip/hip_runtime.h"
#include "foo.cuh"


#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__global__ void foo()
{
    printf("CUDA!\n");
}


void useCUDA()
{

    foo<<<1,25>>>();
    CHECK(hipDeviceSynchronize());

}
